#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <Cuda.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void init_stuff(hiprandState* state)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(1337, idx, 0, &state[idx]);
}

__global__ void make_rand(hiprandState *state, float *randArray) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	randArray[idx] = hiprand_uniform(&state[idx]);
}

void host_function()
{
	int nThreads = 100;
	int nBlocks = 1;
	hiprandState* d_state;
	float* randArray;

	hipMalloc(&d_state, nThreads * nBlocks);
	hipMalloc(&randArray, nThreads * nBlocks);

	init_stuff<<<1, 1>>> (d_state);
	make_rand<<<1, 1>>> (d_state, randArray);
	hipFree(d_state);
}